
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define NX 20000
#define NY 20000

float data[2][NY][NX];

double time_diff_sec(struct timeval st, struct timeval et)
{
    return (double)(et.tv_sec-st.tv_sec)+(et.tv_usec-st.tv_usec)/1000000.0;
}

void init()
{
    int x, y;
    int cx = NX/2, cy = 0; /* center of ink */
    int rad = (NX+NY)/8; /* radius of ink */
    
    for(y = 0; y < NY; y++) {
        for(x = 0; x < NX; x++) {
            float v = 0.0;
            if (((x-cx)*(x-cx)+(y-cy)*(y-cy)) < rad*rad) {
                v = 1.0;
            }
            data[0][y][x] = v;
            data[1][y][x] = v;
        }
    }
    return;
}

/* Calculate for one time step */
/* Input: data[t%2], Output: data[(t+1)%2] */
void calc(int nt)
{
    int t, x, y;

    for (t = 0; t < nt; t++) {
        int from = t%2;
        int to = (t+1)%2;

#if 1
        printf("step %d\n", t);
        //fflush(0);
#endif
    
        for (y = 1; y < NY-1; y++) {
            for (x = 1; x < NX-1; x++) {
                data[to][y][x] = 0.2 * (data[from][y][x]
                                        + data[from][y][x-1]
                                        + data[from][y][x+1]
                                        + data[from][y-1][x]
                                        + data[from][y+1][x]);
            }
        }
    }

    return;
}

int  main(int argc, char *argv[])
{
    struct timeval t1, t2;
    int nt = 20; /* number of time steps */
  
    if (argc >= 2) { /* if an argument is specified */
        nt = atoi(argv[1]);
    }

    init();

    gettimeofday(&t1, NULL);

    calc(nt);

    gettimeofday(&t2, NULL);

    {
        double sec;
        double gflops;
        int op_per_point = 5; // 4 add & 1 multiply per point

        sec = time_diff_sec(t1, t2);
        printf("Elapsed time: %.3lf sec\n", sec);
        gflops = ((double)NX*NY*nt*op_per_point)/sec/1e+9;
        printf("Speed: %.3lf GFlops\n", gflops);
    }

    return 0;
}
