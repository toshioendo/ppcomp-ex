#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


int m;
int n;
int k;
double *A;
double *B;
double *C;

double *DA;
double *DB;
double *DC;

#define BS (64)

double time_diff_sec(struct timeval st, struct timeval et)
{
    return (double)(et.tv_sec-st.tv_sec)+(et.tv_usec-st.tv_usec)/1000000.0;
}

__global__ void matmul_kernel(double *DA, double *DB, double *DC, int m, int n, int k)
{
    int i, j, l;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= m) return; // do nothing

    int lda = m;
    int ldb = k;
    int ldc = m;

    for (j = 0; j < n; j++) {
        double cij = DC[i+j*ldc];
        for (l = 0; l < k; l++) {
            cij += DA[i+l*lda] * DB[l+j*ldb];
        }
        DC[i+j*ldc] = cij;
    }
}

int matmul()
{
    /* invoking (about) m threads */
    matmul_kernel<<<(m+BS-1)/BS, BS>>>(DA, DB, DC, m, n, k);
    return 0;
}

int main(int argc, char *argv[])
{
    int i, j;
    hipError_t rc;

    if (argc < 4) {
        printf("Specify M, N, K.\n");
        exit(1);
    }

    m = atoi(argv[1]);
    n = atoi(argv[2]);
    k = atoi(argv[3]);

    /* allocate matrix region */
    A = (double *)malloc(sizeof(double)*m*k);
    B = (double *)malloc(sizeof(double)*k*n);
    C = (double *)malloc(sizeof(double)*m*n);

    /* setup matrix (column major) */
    /* A is m*k matrix */
    for (j = 0; j < k; j++) {
        for (i = 0; i < m; i++) {
            A[i+j*m] = 1.0;
        }
    }
    /* B is k*n matrix */
    for (j = 0; j < n; j++) {
        for (i = 0; i < k; i++) {
            B[i+j*k] = 10.0;
        }
    }
    /* C is m*n matrix */
    for (j = 0; j < n; j++) {
        for (i = 0; i < m; i++) {
            C[i+j*m] = 0.0;
        }
    }

    /* allocate device memory */
    rc = hipMalloc((void**)&DA, sizeof(double)*m*k);
    if (rc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed\n"); exit(1);
    }
    rc = hipMalloc((void**)&DB, sizeof(double)*k*n);
    if (rc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed\n"); exit(1);
    }
    rc = hipMalloc((void**)&DC, sizeof(double)*m*n);
    if (rc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed\n"); exit(1);
    }

    /* Repeat same computation */
    for (i = 0; i < 5; i++) {
        struct timeval st, st2, et2, et;
        long flop;
        double insec, compsec, outsec, sec;

        gettimeofday(&st, NULL);

        /* copy input matrices from host to device */
        hipMemcpy(DA, A, sizeof(double)*m*k, hipMemcpyDefault);
        hipMemcpy(DB, B, sizeof(double)*k*n, hipMemcpyDefault);
        hipMemcpy(DC, C, sizeof(double)*m*n, hipMemcpyDefault);
        hipDeviceSynchronize(); /* for precise time measurement */

        gettimeofday(&st2, NULL);

        /* computation */
        matmul();
        hipDeviceSynchronize(); /* for precise time measurement */

        gettimeofday(&et2, NULL);

        /* copy output matrices from device to host */
        hipMemcpy(C, DC, sizeof(double)*m*n, hipMemcpyDefault);
        hipDeviceSynchronize(); /* for precise time measurement */

        gettimeofday(&et, NULL);

        flop = 2.0*(double)m*(double)n*(double)k;
        insec = time_diff_sec(st, st2);
        compsec = time_diff_sec(st2, et2);
        outsec = time_diff_sec(et2, et);
        sec = time_diff_sec(st, et); // insec + compsec + outsec;
        printf("copyin: %lf sec, compute: %lf sec, copyout: %lf sec\n",
               insec, compsec, outsec);
        printf("Matmul took %lf sec --> %.3lf GFlops  (with data transfer)\n",
               sec, (double)flop/(double)sec/1e+9);
        printf("            %lf sec --> %.3lf GFlops  (without data transfer)\n",
               compsec, (double)flop/(double)compsec/1e+9);
    }

    hipFree(DA);
    hipFree(DB);
    hipFree(DC);
    free(A);
    free(B);
    free(C);
    return 0;
}
